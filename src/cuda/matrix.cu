

#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/driver_types.h>
struct CMatrix{
    int rows;
    int cols;
    const float *data;
    int len;
};

__global__ void matrixMul(float *result, const float *matrix_a, const float *matrix_b,  int rowsA, int colsA, int colsB){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < rowsA && col < colsB){
        float res = 0.0f;
        for(int i = 0; i < colsA; i++){
            res += matrix_a[row * colsA + i] * matrix_b[i * colsB + col];
        }
        result[row * colsB + col] = res;
    }
}
__global__ void matrixAdd(float *result, const float *matrix_a, const float *matrix_b, int rowsA, int colsA){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < rowsA && col < colsA){
        float res = 0.0f;
        res += matrix_a[row * colsA + col] + matrix_b[row * colsA + col];
        result[row * colsA + col] = res;
    }
}
extern "C" {
    void cuda_matrix_mul(float *result, const CMatrix *matrix_a, const CMatrix *matrix_b){
        //Transfer to device mem :)
        float *result_dev, *matrix_a_dev, *matrix_b_dev;
        hipMalloc((void**)&result_dev, matrix_a->rows * matrix_b->cols * sizeof(float));

        hipMalloc((void**)&matrix_a_dev, matrix_a->len*sizeof(float));
        hipMalloc((void**)&matrix_b_dev, matrix_b->len*sizeof(float));

        //Copy matrix A and B to A-device and B-device memory allocations
        hipMemcpy(matrix_a_dev, matrix_a->data, matrix_a->len * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(matrix_b_dev, matrix_b->data, matrix_b->len * sizeof(float), hipMemcpyHostToDevice);

        //Split our grids and block sizes into blocks of 16x16
        dim3 blockSize(16,16);
        dim3 gridSize((matrix_b->cols + blockSize.x - 1) / blockSize.x, (matrix_a->rows + blockSize.y - 1) / blockSize.y);
        //Call cuda function!
        matrixMul<<<gridSize, blockSize>>>(result_dev, matrix_a_dev, matrix_b_dev, matrix_a->rows, matrix_a->cols, matrix_b->cols); 

        hipError_t err = hipMemcpy(result, result_dev, matrix_a->rows * matrix_b->cols * sizeof(float), hipMemcpyDeviceToHost);
        if(err != hipSuccess){
            printf("Big ass cuda error: %s\n", hipGetErrorString(err));
        }
        //Memory management :D
        hipFree(result_dev);
        hipFree(matrix_a_dev);
        hipFree(matrix_b_dev);
    }
    void cuda_matrix_add(float *result, const CMatrix *matrix_a, const CMatrix *matrix_b){
        //Transfer to device mem :)
        float *result_dev, *matrix_a_dev, *matrix_b_dev;
        hipMalloc((void**)&result_dev, matrix_a->len * sizeof(float));

        hipMalloc((void**)&matrix_a_dev, matrix_a->len*sizeof(float));
        hipMalloc((void**)&matrix_b_dev, matrix_b->len*sizeof(float));

        //Copy matrix A and B to A-device and B-device memory allocations
        hipMemcpy(matrix_a_dev, matrix_a->data, matrix_a->len * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(matrix_b_dev, matrix_b->data, matrix_b->len * sizeof(float), hipMemcpyHostToDevice);

        //Split our grids and block sizes into blocks of 16x16
        dim3 blockSize(16,16);
        dim3 gridSize((matrix_b->cols + blockSize.x - 1) / blockSize.x, (matrix_a->rows + blockSize.y - 1) / blockSize.y);
        //Call cuda function!
        matrixAdd<<<gridSize, blockSize>>>(result_dev, matrix_a_dev, matrix_b_dev, matrix_a->rows, matrix_a->cols); 

        hipMemcpy(result, result_dev, matrix_a->len * sizeof(float), hipMemcpyDeviceToHost);
        //Memory management :D
        hipFree(result_dev);
        hipFree(matrix_a_dev);
        hipFree(matrix_b_dev);
    }
}
/*
int main() {
    //Test matrix math
    int rowsA = 3;
    int colsA = 4;
    int rowsB = 4; // Should match colsA for multiplication
    int colsB = 2;

    float matrixA[rowsA * colsA];
    float matrixB[rowsB * colsB];

    for (int i = 0; i < rowsA * colsA; ++i) {
        matrixA[i] = (float)(rand() % 10); 
        printf("%f ", matrixA[i]);
    }
    printf("\n");
    for (int i = 0; i < rowsB * colsB; ++i) {
        matrixB[i] = (float)(rand() % 10); 
        printf("%f ", matrixB[i]);
    }

    struct CMatrix matA = { rowsA, colsA, matrixA, rowsA * colsA };
    struct CMatrix matB = { rowsB, colsB, matrixB, rowsB * colsB };

    int rowsResult = rowsA;
    int colsResult = colsB;

    float resultMatrix[rowsResult * colsResult];

    struct CMatrix matResult = { rowsResult, colsResult, resultMatrix, rowsResult * colsResult };

    cuda_matrix_mul(resultMatrix, &matA, &matB);

    printf("Result Matrix:\n");
    for (int i = 0; i < rowsResult; ++i) {
        for (int j = 0; j < colsResult; ++j) {
            printf("%.2f ", resultMatrix[i * colsResult + j]);
        }
        printf("\n");
    }

    return 0;
}*/
